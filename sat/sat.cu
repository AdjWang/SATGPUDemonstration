#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <math.h>
#include "sat.h"

// GPU utilities
#include <hip/hip_runtime.h>
#include "dependence.h"

// vector ---------------------------------------------------------------------

static vector_t vector_sub(const vector_t vec1, const vector_t vec2){
    return (vector_t){
        .x = vec1.x - vec2.x,
        .y = vec1.y - vec2.y,
    };
}

static double vector_dot(const vector_t vec1, const vector_t vec2){
    return vec1.x*vec2.x + vec1.y*vec2.y;
}

static double vecotr_magnitude(const vector_t vec){
    return sqrt(vec.x*vec.x + vec.y*vec.y);
}

static vector_t vector_normalize(const vector_t vec){
    double mag = vecotr_magnitude(vec);
    return (vector_t){
        .x = vec.x / mag,
        .y = vec.y / mag,
    };
}

static vector_t vector_perpendicular(const vector_t vec){
    // or (y, -x)
    return (vector_t){
        .x = -vec.y,
        .y = vec.x,
    };
}

// projection -----------------------------------------------------------------

static BOOL projection_is_overlap(const projection_t projection1, const projection_t projection2){
    return (projection1.left < projection2.right
        && projection1.right > projection2.left);
}

// ploygon --------------------------------------------------------------------

polygon_t* new_polygon(int n_vertex){
    if(n_vertex <= 2){
        return NULL;
    }
    polygon_t* polygon = (polygon_t*)malloc(sizeof(polygon_t));
    if(!polygon){
        RAISE("malloc failed\n");
    }
    polygon->n = n_vertex;
    polygon->vertices = (point_t*)malloc(sizeof(point_t)*n_vertex);
    if(!polygon->vertices){
        RAISE("malloc failed\n");
    }
    polygon->axes = (vector_t*)malloc(sizeof(vector_t)*n_vertex);
    if(!polygon->axes){
        RAISE("malloc failed\n");
    }
    return polygon;
}

void del_polygon(polygon_t* polygon){
    if(!polygon){
        return;
    }
    if(polygon->vertices){
        free(polygon->vertices);
        polygon->vertices = NULL;
    }
    if(polygon->axes){
        free(polygon->axes);
        polygon->axes = NULL;
    }
    free(polygon);
}

static int polygon_print_point_list(FILE* stream, const point_t* point_list, int n){
    int c = 0;
    c += fprintf(stream, "[");
    for(int i=0; i<n; i++){
        c += fprintf(stream, "(%.16lf,%.16lf),", point_list[i].x, point_list[i].y);
    }
    c += fprintf(stream, "]\n");
    return c;
}

int polygon_print(FILE* stream, const polygon_t* polygon){
    int c = 0;
    c += fprintf(stream, "%d\n", polygon->n);
    c += polygon_print_point_list(stream, polygon->vertices, polygon->n);
    return c;
}

static void polygon_get_axes(polygon_t* polygon){
    for(int i=0; i<polygon->n; i++){
        point_t p1 = polygon->vertices[i];
        point_t p2 = polygon->vertices[(i+1) == polygon->n ? 0 : (i+1)];
        vector_t edge = vector_sub(*(vector_t*)&p1, *(vector_t*)&p2);
        vector_t norm = vector_normalize(edge);
        vector_t perp = vector_perpendicular(norm);
        polygon->axes[i] = perp;    // copy by value
    }
}

static projection_t polygon_project(const polygon_t* polygon, const vector_t axis){
    double proj_min=INFINITY, proj_max=-INFINITY;
    for(int i=0; i<polygon->n; i++){
        double proj_num = vector_dot(axis, *(vector_t*)&(polygon->vertices[i]));
        if(proj_num < proj_min){
            proj_min = proj_num;
        }
        if(proj_num > proj_max){
            proj_max = proj_num;
        }
    }
    return (projection_t){
        .left = proj_min,
        .right = proj_max,
    };
}

BOOL polygon_is_overlap(const polygon_t* polygon1, const polygon_t* polygon2){
    // loop over the axes1
    vector_t* axes1 = polygon1->axes;
    for(int i=0; i<polygon1->n; i++){
        vector_t axis = axes1[i];
        // project both shapes onto the axis
        projection_t p1 = polygon_project(polygon1, axis);
        projection_t p2 = polygon_project(polygon2, axis);
        // do the projections overlap?
        if (!projection_is_overlap(p1, p2)) {
            // then we can guarantee that the shapes do not overlap
            return FALSE;
        }
    }

    // loop over the axes2
    vector_t* axes2 = polygon2->axes;
    for(int i=0; i<polygon2->n; i++){
        vector_t axis = axes2[i];
        // project both shapes onto the axis
        projection_t p1 = polygon_project(polygon1, axis);
        projection_t p2 = polygon_project(polygon2, axis);
        // do the projections overlap?
        if (!projection_is_overlap(p1, p2)) {
            // then we can guarantee that the shapes do not overlap
            return FALSE;
        }
    }

    return TRUE;
}

// cpu implemention -----------------------------------------------------------

void detect_overlap(polygon_t** polygon_list, int** result, int n){
    ASSERT(result != NULL, "invalid result: NULL\n");
    ASSERT(n > 0, "empty list: n <= 0\n");

    // calculate axes
    for(int i=0; i<n; i++){
        polygon_t* polygon = polygon_list[i];
        polygon_get_axes(polygon);
    }

    // calculate overlap
    for(int i=0; i<n-1; i++){
        for(int j=i+1; j<n; j++){
            if(polygon_is_overlap(polygon_list[i], polygon_list[j])){
                result[i][j] = result[j][i] = 1;
            }
        }
    }
}

// gpu implemention -----------------------------------------------------------

// device utilities

__device__ static vector_t vector_sub_gpu(const vector_t vec1, const vector_t vec2){
    return (vector_t){
        .x = vec1.x - vec2.x,
        .y = vec1.y - vec2.y,
    };
}

__device__ static double vector_dot_gpu(const vector_t vec1, const vector_t vec2){
    return vec1.x*vec2.x + vec1.y*vec2.y;
}

__device__ static double vecotr_magnitude_gpu(const vector_t vec){
    return sqrt(vec.x*vec.x + vec.y*vec.y);
}

__device__ static vector_t vector_normalize_gpu(const vector_t vec){
    double mag = vecotr_magnitude_gpu(vec);
    return (vector_t){
        .x = vec.x / mag,
        .y = vec.y / mag,
    };
}

__device__ static vector_t vector_perpendicular_gpu(const vector_t vec){
    // or (y, -x)
    return (vector_t){
        .x = -vec.y,
        .y = vec.x,
    };
}

// Get minimal and maximal value of endpoints.
__device__ static projection_t min_max_gpu(const double* endpoints, int n){
    double proj_min=INFINITY, proj_max=-INFINITY;
    // printf("endpoings: [%d, %d)\n", start, end);
    for(int i=0; i<n; i++){
        double proj_num = endpoints[i];
        // printf("%lf ", proj_num);
        if(proj_num < proj_min){
            proj_min = proj_num;
        }
        if(proj_num > proj_max){
            proj_max = proj_num;
        }
    }
    // printf("\n");
    return (projection_t){
        .left = proj_min,
        .right = proj_max,
    };
}

__device__ static BOOL projection_is_overlap_gpu(const projection_t* projection1, const projection_t* projection2){
    return (projection1->left < projection2->right
        && projection1->right > projection2->left);
}

// Flatten vertices of polygons for further parallelizing.
// description:
//   1. vertices of polygon i is vertices[i_polygon_map[i] : i_polygon_map[i] + polygon_n_map[i]]
//   2. owner_map is the flattened indices.
//   e.g. polygons(a triangle and a regtangle) whose number of vertices is {3, 4}, its:
//        vertices is {point0, point1, point2, point3, point4, point5, point6}
//        i_polygon_map is {0, 3}
//        polygon_n_map is {3, 4}
//        owner_map is {0, 0, 0, 1, 1, 1, 1}
static int util_flatten(polygon_t** polygon_list, int n_polygon,
                        /*out*/int** p_i_polygon_map_gpu, /*out*/int** p_polygon_n_map_gpu,
                        /*out*/int** p_owner_map_gpu, /*out*/point_t** p_vertices_gpu){
    ASSERT(*p_i_polygon_map_gpu == NULL, "*p_i_polygon_map_gpu should be NULL\n");
    ASSERT(*p_polygon_n_map_gpu == NULL, "*p_polygon_n_map_gpu should be NULL\n");
    ASSERT(*p_owner_map_gpu == NULL, "*p_owner_map_gpu should be NULL\n");
    ASSERT(*p_vertices_gpu == NULL, "*p_vertices_gpu should be NULL\n");

    // allocate local temp buf
    int* i_polygon_map = (int*)malloc(n_polygon * sizeof(int));
    if(!i_polygon_map){
        RAISE("malloc failed.\n");
    }
    int* polygon_n_map = (int*)malloc(n_polygon * sizeof(int));
    if(!polygon_n_map){
        RAISE("malloc failed.\n");
    }

    // get n_vertex to flatten vertices
    int n_vertex = 0;
    for(int i=0; i<n_polygon; i++){
        // prefix sum
        i_polygon_map[i] = n_vertex;

        int polygon_n_vertex = polygon_list[i]->n;
        // just every n
        polygon_n_map[i] = polygon_n_vertex;
        // count
        n_vertex += polygon_n_vertex;
    }

    CHECK(hipMalloc(p_i_polygon_map_gpu, n_polygon * sizeof(int)));
    CHECK(hipMemcpy(*p_i_polygon_map_gpu, i_polygon_map, n_polygon * sizeof(int), hipMemcpyHostToDevice));

    CHECK(hipMalloc(p_polygon_n_map_gpu, n_polygon * sizeof(int)));
    CHECK(hipMemcpy(*p_polygon_n_map_gpu, polygon_n_map, n_polygon * sizeof(int), hipMemcpyHostToDevice));

    free(polygon_n_map);
    free(i_polygon_map);

    int* owner_map = (int*)malloc(n_vertex * sizeof(int));
    if(!owner_map){
        RAISE("malloc failed.\n");
    }

    int* p_owner = owner_map;
    for(int i=0; i<n_polygon; i++){
        int polygon_n_vertex = polygon_list[i]->n;
        for(int j=0; j<polygon_n_vertex; j++){
            *p_owner = i;
            p_owner++;
        }
    }

    CHECK(hipMalloc(p_owner_map_gpu, n_vertex * sizeof(int)));
    CHECK(hipMemcpy(*p_owner_map_gpu, owner_map, n_vertex * sizeof(int), hipMemcpyHostToDevice));

    free(owner_map);

    // alloc point_t vertices_gpu[] on device
    CHECK(hipMalloc(p_vertices_gpu, n_vertex * sizeof(point_t)));

    // copy vertices from host to device
    point_t* p_vertex = *p_vertices_gpu;
    for(int i=0; i<n_polygon; i++){
        int polygon_n_vertex = polygon_list[i]->n;
        point_t* vertices = polygon_list[i]->vertices;
        CHECK(hipMemcpy(p_vertex, vertices, polygon_n_vertex * sizeof(point_t), hipMemcpyHostToDevice));
        p_vertex += polygon_n_vertex;
    }

    return n_vertex;
}

// kernel functions

// Calculate axes of polygons.
// nx = n_vertex, ny = 1
// input:
//   point_t vertices[n_vertex]
//   int n_vertex
//   int owner_map_gpu[n_vertex]
//   int polygon_n_map_gpu[n_vertex]
// output:
//   vector_t axes[n_vertex]
__global__ static void kernel_get_axis(point_t* vertices, int n_vertex, int* owner_map_gpu, int* polygon_n_map_gpu, /*out*/vector_t* axes){
    int ix = threadIdx.x + blockIdx.x * blockDim.x;

    if(ix < n_vertex){
        // e.g.
        // ix:              0, 1, 2, 3, 4, 5, 6
        // owmer_map:       0, 0, 0, 1, 1, 1, 1
        // polygon_n_map:   3, 4
        // i_vertex_a:      0, 1, 2, 3, 4, 5, 6
        // i_vertex_b:      1, 2, 0, 4, 5, 6, 3

        // get ia and ib
        int i_vertex_a = ix;
        int i_vertex_b = (ix + 1) >= n_vertex ? 0 : (ix + 1);
        if(owner_map_gpu[i_vertex_a] != owner_map_gpu[i_vertex_b]){
            int i_polygon_a = owner_map_gpu[i_vertex_a];
            int polygon_n_vertex = polygon_n_map_gpu[i_polygon_a];
            i_vertex_b = i_vertex_a + 1 - polygon_n_vertex;
        }
        
        // calculate axis
        point_t p1 = vertices[i_vertex_a];
        point_t p2 = vertices[i_vertex_b];
        vector_t edge = vector_sub_gpu(*(vector_t*)&p1, *(vector_t*)&p2);
        vector_t norm = vector_normalize_gpu(edge);
        vector_t perp = vector_perpendicular_gpu(norm);

        axes[i_vertex_a] = perp;
    }
}
void calculate_axes(point_t* vertices_gpu, int n_vertex, int* owner_map_gpu, int* polygon_n_map_gpu,
                    /*out*/vector_t** p_axes_gpu){
    ASSERT(*p_axes_gpu == NULL, "*p_axes_gpu should be NULL\n");

    // calculate on the device
    CHECK(hipMalloc(p_axes_gpu, n_vertex * sizeof(vector_t)));

    const int dimx = 128;
    dim3 block(dimx);
    dim3 grid((n_vertex - 1)/block.x + 1);

    kernel_get_axis<<<grid, block>>>(vertices_gpu, n_vertex, owner_map_gpu, polygon_n_map_gpu, *p_axes_gpu);
    CHECK(hipGetLastError());
    CHECK(hipDeviceSynchronize());
}

// Dot to get every single projection endpoint.
// nx = n_vertex, ny = n_vertex
// input:
//   point_t vertices[n_vertex]
//   vector_t axes[n_vertex]
//   int owner_map[n_vertex]
//   int n_vertices
// output:
//   double projection_endpoint[n_vertex * n_vertex]
//     actually should be projection_endpoints[n_axis][n_vertex]
__global__ static void kernel_get_projection_endpoints(point_t* vertices, vector_t* axes, int* owner_map, int n_vertex,
                                                       /*out*/double* projection_endpoints){
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    // int idx = iy * gridDim.x * blockDim.x + ix;

    // printf("thread_id(%d,%d) block_id(%d,%d) coordinate(%d,%d) global index %2d\n", \
    //     threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y, ix, iy, idx);
    int i_vertex = ix;
    int i_axis = iy;
    int i_proj = i_axis * n_vertex + i_vertex;
    if(i_vertex < n_vertex && i_axis < n_vertex){
        // if(owner_map[i_vertex] != owner_map[i_axis]){
            double projection = vector_dot_gpu(*(vector_t*)&vertices[i_vertex], axes[i_axis]);
            projection_endpoints[i_proj] = projection;
        // }
    }
}
void calculate_projection_endpoints(point_t* vertices_gpu, vector_t* axes_gpu, int* owner_map_gpu, int n_vertex,
                                           /*out*/double** p_projection_endpoints_gpu){
    ASSERT(*p_projection_endpoints_gpu == NULL, "*p_projection_endpoint should be NULL\n");

    // allocate projection return value on the device
    // all projection endpoints of vertices and axes, not [min, max] segment
    CHECK(hipMalloc(p_projection_endpoints_gpu, n_vertex*n_vertex * sizeof(double)));

    const int dimx = 32;
    const int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((n_vertex - 1)/block.x + 1, (n_vertex - 1)/block.y + 1);

    // printf("%d\n", n_vertex);
    kernel_get_projection_endpoints<<<grid, block>>>(vertices_gpu, axes_gpu, owner_map_gpu, n_vertex, /*out*/*p_projection_endpoints_gpu);
    CHECK(hipGetLastError());
    CHECK(hipDeviceSynchronize());
}

// Aggregate projection endpoints to projections([min, max]).
// nx = n_vertex, ny = n_polygon
// input:
//   double projection_endpoints[n_vertex * n_vertex]
//     actually should be projection_endpoints[n_axis][n_vertex]
//   int i_polygon_map[n_polygon]
//     map i_polygon to the start index of vertices.
//   int polygon_n_map[n_vertex]
//     get polygon_n_vertex of the i_vertex of vertices.
//   int n_vertex
//   int n_polygon
// output:
//   projection_t projection_map[n_polygon * n_vertex]
//     actually should be projection_map[n_vertex][n_polygon]
// description:
//   1. for a certain i_polygon, its vertices is
//      projection_endpoints[i_axis][i_polygon_map[i_polygon] : i_polygon_map[i_polygon] + polygon_n_vertex[i_polygon]]
__global__ static void kernel_get_projection_segments(double* projection_endpoints, int* i_polygon_map, int* polygon_n_map,
                                             int n_vertex, int n_polygon, /*out*/projection_t* projection_map){
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    // int idx = iy * gridDim.x * blockDim.x + ix;

    // printf("thread_id(%d,%d) block_id(%d,%d) coordinate(%d,%d) global index %2d\n", \
    //     threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y, ix, iy, idx);

    int i_axis = ix;
    int i_polygon = iy;
    if(i_axis < n_vertex && i_polygon < n_polygon){
        // index of a polygon in flattened vertices
        int polygon_idx = i_polygon_map[i_polygon];
        // n vertices of a polygon
        int polygon_n_vertex = polygon_n_map[i_polygon];
        // calculate projection segment
        double* vertices_slice = &projection_endpoints[i_axis*n_vertex + polygon_idx];
        projection_t projection_segment = min_max_gpu(vertices_slice, polygon_n_vertex);
        // return
        projection_map[i_axis*n_polygon + i_polygon] = projection_segment;
    }
}
void calculate_projection_segments(double* projection_endpoints_gpu, int* i_polygon_map_gpu, int* polygon_n_map_gpu,
                                   int n_vertex, int n_polygon, /*out*/projection_t** p_projection_map){
    ASSERT(*p_projection_map == NULL, "*p_projection_map should be NULL\n");

    CHECK(hipMalloc(p_projection_map, n_vertex*n_polygon * sizeof(projection_t)));

    const int dimx = 32;
    const int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((n_vertex - 1)/block.x + 1, (n_polygon - 1)/block.y + 1);

    kernel_get_projection_segments<<<grid, block>>>(projection_endpoints_gpu, i_polygon_map_gpu, polygon_n_map_gpu, n_vertex, n_polygon, *p_projection_map);
    CHECK(hipGetLastError());
    CHECK(hipDeviceSynchronize());
}

// Get overlapping map of polygons.
// nx = n_polygon, ny = n_polygon, nz = n_vertex
// input:
//   projection_t projection_map[n_vertex * n_polygon]
//     actually should be projection_map[n_vertex][n_polygon]
//   int owner_map[n_vertex]
// output:
//   int result[n_polygon * n_polygon]
// description:
//   check if polygon at ix and at iy overlapping on i_axis(iz)
__global__ static void kernel_get_overlapping(projection_t* projection_map, int n_vertex, int n_polygon, /*out*/int* result){
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    int iz = threadIdx.z + blockIdx.z * blockDim.z;

    int i_polygon_a = ix;
    int i_polygon_b = iy;
    int i_axis = iz;
    if(i_polygon_a < n_polygon && i_polygon_b < n_polygon && i_axis < n_vertex){
        if(result[i_polygon_a*n_polygon + i_polygon_b] != 0){
            projection_t* p_projection_a = &projection_map[i_axis*n_polygon + i_polygon_a];
            projection_t* p_projection_b = &projection_map[i_axis*n_polygon + i_polygon_b];
            if(projection_is_overlap_gpu(p_projection_a, p_projection_b) == FALSE){
                // NOTE: test race conditions
                result[i_polygon_a*n_polygon + i_polygon_b] &= 0;   // set not overlapping
            }
        }// else the 2 polygons can not be overlapped, no need to calculate
    }
}
void calculate_is_overlapping(projection_t* projection_map_gpu, int n_vertex, int n_polygon, /*out*/int** p_result){
    ASSERT(*p_result == NULL, "*p_result should be NULL\n");

    CHECK(hipMalloc(p_result, n_polygon*n_polygon * sizeof(int)));
    CHECK(hipMemset(*p_result, 0xFF, n_polygon*n_polygon * sizeof(int)));

    const int dimx = 4;
    const int dimy = 4;
    const int dimz = 32;
    dim3 block(dimx, dimy, dimz);
    dim3 grid((n_polygon - 1)/block.x + 1, (n_polygon - 1)/block.y + 1, (n_vertex - 1)/block.z + 1);
    // printf("grid (%d, %d, %d)\n", grid.x, grid.y, grid.z);

    kernel_get_overlapping<<<grid, block>>>(projection_map_gpu, n_vertex, n_polygon, *p_result);
    CHECK(hipGetLastError());
    CHECK(hipDeviceSynchronize());
}

// global interfaces and export for user

void detect_overlap_gpu(polygon_t** polygon_list, int** result, int n_polygon){
    ASSERT(result != NULL, "result should not be NULL\n");

    // 1. flatten vertices of all polygons to device
    point_t* vertices_gpu = NULL;   // destructor: hipFree
    // vertices of polygon i is vertices[i_polygon_map[i] : i_polygon_map[i] + polygon_n_map[i]]
    int* i_polygon_map_gpu = NULL;   // destructor: hipFree
    int* polygon_n_map_gpu = NULL;   // destructor: hipFree
    int* owner_map_gpu = NULL;   // destructor: hipFree
    int n_vertex = util_flatten(polygon_list, n_polygon, /*out*/&i_polygon_map_gpu, /*out*/&polygon_n_map_gpu, /*out*/&owner_map_gpu, /*out*/&vertices_gpu);

    // 2. calculate axes
    vector_t* axes_gpu = NULL;   // destructor: hipFree
    calculate_axes(vertices_gpu, n_vertex, owner_map_gpu, polygon_n_map_gpu, /*out*/&axes_gpu);

    // 3. calculate projection endpoints
    double* projection_endpoints_gpu = NULL;   // destructor: hipFree
    calculate_projection_endpoints(vertices_gpu, axes_gpu, owner_map_gpu, n_vertex, /*out*/&projection_endpoints_gpu);

    // 4. calculate projection segments
    projection_t* projection_map_gpu = NULL;   // destructor: hipFree
    calculate_projection_segments(projection_endpoints_gpu, i_polygon_map_gpu, polygon_n_map_gpu, n_vertex, n_polygon, /*out*/&projection_map_gpu);

    // 5. calculate overlapping
    int* result_gpu = NULL;
    calculate_is_overlapping(projection_map_gpu, n_vertex, n_polygon, /*out*/&result_gpu);

    // set result
    int* result_host = (int*)malloc(n_polygon*n_polygon * sizeof(int));
    if(!result_host){
        RAISE("malloc failed.\n");
    }
    CHECK(hipMemcpy(result_host, result_gpu, n_polygon*n_polygon * sizeof(int), hipMemcpyDeviceToHost));
    for(int i_polygon_a=0; i_polygon_a<n_polygon; i_polygon_a++){
        for(int i_polygon_b=i_polygon_a; i_polygon_b<n_polygon; i_polygon_b++){
            if(i_polygon_a == i_polygon_b){
                result[i_polygon_a][i_polygon_b] = 0;
                continue;
            }
            int i = i_polygon_a*n_polygon + i_polygon_b;
            result[i_polygon_a][i_polygon_b] = (result_host[i] != 0);
            result[i_polygon_b][i_polygon_a] = (result_host[i] != 0);
        }
    }
    free(result_host);
    
    // recycle resources
    // from 5.
    hipFree(result_gpu);
    // from 4.
    hipFree(projection_map_gpu);
    // from 3.
    hipFree(projection_endpoints_gpu);
    // from 2.
    hipFree(axes_gpu);
    // from 1.
    hipFree(owner_map_gpu);
    hipFree(polygon_n_map_gpu);
    hipFree(i_polygon_map_gpu);
    hipFree(vertices_gpu);

    // typedef unsigned long long int uint64_t;
    // uint64_t nv = (uint64_t)n_vertex;
    // uint64_t np = (uint64_t)n_polygon;
    // uint64_t n_kernel = max(nv*nv, np*np*nv);
    // printf("kernel num: %llu\n", n_kernel);
}
